#include "hip/hip_runtime.h"
//
// Created by Derek on 2022/11/16.
//
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include "classifier.h"

using namespace std;

typedef vector<vector<float>> vvf;


void split(const std::string &s, std::vector<std::string> &v, const std::string &c) {
    v.clear();
    std::string::size_type pos1, pos2;
    pos2 = s.find(c);
    pos1 = 0;
    while (std::string::npos != pos2) {
        v.push_back(s.substr(pos1, pos2 - pos1));
        pos1 = pos2 + c.size();
        pos2 = s.find(c, pos1);
    }
    if (pos1 != s.length())
        v.push_back(s.substr(pos1));
}


void append_img_to_vec(const string &filename, vector<vector<float>> &data, vector<int> &label_set, int label) {
    ifstream in(filename, ios::in);
    string line;
    while (getline(in, line)) {
        vector<string> strs;
        split(line, strs, " ");
        vector<float> temp;
        for (auto &str: strs) {
            if (str == "\r" || str == "\r\n" || str == "\n") continue;
            temp.push_back(stof(str));
        }
        data.push_back(temp);
        label_set.emplace_back(label);
    }
}

#define RUNNING_TIMES 10

void run_test(const string& test_name, int h, int w, int eval_batch=10, const string& res_path = "res.csv") {
    ofstream out(res_path, ios::app);

    // write test name
    out << test_name << endl;

    // load data and run test
    vvf train_data;
    vecI train_label;
    append_img_to_vec(test_name + "/0_train.txt", train_data, train_label, 1);
    append_img_to_vec(test_name + "/1_train.txt", train_data, train_label, -1);

    printf("data_size: %d; data_width: %d\n", (int) train_data.size(), (int) train_data[0].size());
    printf("label_size: %d\n", (int) train_label.size());

    vvf test_data;
    vecI test_label;
    append_img_to_vec(test_name + "/0_test.txt", test_data, test_label, 1);
    append_img_to_vec(test_name + "/1_test.txt", test_data, test_label, -1);
    printf("test_data_size: %d; test_data_width: %d\n", (int) test_data.size(), (int) test_data[0].size());
    printf("test_label_size: %d\n", (int) test_label.size());

    for (int i = 0; i < RUNNING_TIMES; i++) {
        auto classifier = BinaryClassifier(train_data, train_label, test_data, test_label, h, w);
        classifier.eval_batch = eval_batch;
        classifier.generations = 50;
        classifier.init();

        // train
        auto start_time = clock();
        classifier.train();
        auto dur = (float) (clock() - start_time) / (float) CLOCKS_PER_SEC;
        cout << "training time: " << dur << "s" << endl;

        // test
        classifier.run_test();
        cout << endl;

        // write training time and test accuracy
        out << dur << "," << classifier.best_test_program.fitness << endl;
    }
}


int main(int argc, const char *argv[]) {
    run_test("../jaffe", 128, 128, 500, "../temp.csv");
//    run_test("cifar", 32, 32, 10);
//    run_test("mnist", 28, 28, 10);
//    run_test("coil", 128, 128, 500);
//    run_test("kth", 128, 128, 100);
//    run_test("uiuc", 40, 100, 250);
    return 0;
}